#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__global__ void increment(int *data_ptr)
{
  (*data_ptr)++;
}

__global__ void at_increment(int *data_ptr)
{
  atomicAdd(data_ptr, 1);
}
 
int main(void)
{
  thrust::device_vector<int> data_ptr(1);
  data_ptr[0] = 0;
  increment<<<100,100>>>(thrust::raw_pointer_cast(&data_ptr[0]));
  //at_increment<<<100,100>>>(thrust::raw_pointer_cast(&data_ptr[0]));
  hipDeviceSynchronize();
  std::cout<<"Actual "<<data_ptr[0]<<std::endl;
  std::cout<<"Expected "<<10*1000<<std::endl;

  return 0;
}
